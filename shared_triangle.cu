#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "mmio.h"
#include "mmio.c"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/random.h>
#include <thrust/sort.h>
#include <time.h>


#define CUDA_CALL(x)                                                           \
  {                                                                            \
    if ((x) != hipSuccess) {                                                  \
      printf("CUDA error at %s:%d\n", __FILE__, __LINE__);                     \
      printf("  %s\n", hipGetErrorString(hipGetLastError()));                \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }

__global__ void findColStart(int *dJ, int len, int *col_ptr) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i<len){
  if (i > 0 ) {
    if (dJ[i] != dJ[i - 1]) {
      col_ptr[dJ[i]] = i;
    }
  } else {
    col_ptr[dJ[0]] = 0;
  }}


}

__global__ void InitColStart(int len, int *col_ptr) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i<len){
        col_ptr[i]=-1;

  }
}

#define sharedsize 65
__global__ void computeRow1(int* dI,int* dJ,int nz,int* col_ptr,int* out, int N) {
int i=blockIdx.x;
while(i<N){
    //if(threadIdx.x==0 && blockIdx.x==0){
    //printf("blockIdx=%d\n",blockIdx.x );}
    __shared__ int blockCol[sharedsize];//len of column
    __shared__ int nt[sharedsize];  
    int colStart=col_ptr[i];
    int tid=threadIdx.x;
    if(colStart<0){
        if(tid<32){
        out[i]=0;}
        return;
    }

  
    int j=tid;
    //if(threadIdx.x==0&& blockIdx.x ==0){
    //printf("threadIdx=%d\n",threadIdx.x );}
    while(j<sharedsize)
    {   
        if(dJ[j+colStart]==i){
            blockCol[j]=dI[j+colStart];
            }
        else{
            blockCol[j]=0;
        }
        j+=blockDim.x;
    }
      __syncthreads();
   // if(threadIdx.x==0&& i ==0){
   //     for(int i=0;i<sharedsize;i++){
    //        printf("%d-",blockCol[i] );
    //    }
   //     printf("\n");
    //    }

  
     j=tid;
     int k1;
     int k2;
     int s;
     while(j<sharedsize)
    {   s=0;
        k1=0;
        int x=blockCol[j];
        k2=col_ptr[x];
        int r1;
        int r2;
        if(k2>0){
            r1=blockCol[k1];
            r2=dI[k2];
            while(r1>0 && dJ[k2]==x ) {
            //    if(threadIdx.x==0&& i ==0){
            //       printf("r1=%d,r2=%d\n",r1,r2 );
            //        printf("k1=%d,k2=%d\n",k1,k2 );
            //    }
                if(r1==r2){
                    s++;      
                    k1++;
                    k2++;

                }else if(r1>r2){
                    k2++;

                }else{
                k1++;
                }
                if(k2==nz){break;}
                r1=blockCol[k1];
                r2=dI[k2];
            }
        }

        nt[j]=s;
        j+=blockDim.x;
    }
    __syncthreads();
    j=tid+blockDim.x;
    while(j<sharedsize){
    nt[tid]=nt[tid]+nt[j];
    j+=blockDim.x;
    }
    __syncthreads();


    //do reduction in shared mem 
    for( s=blockDim.x/2; s>0;s>>=1){
        if(tid<s){
        nt[tid]+=nt[tid+s];
        }
    __syncthreads();
    }

    if(tid<32){
    out[i]=nt[0];}
  
    i+=gridDim.x;
  }
}



int main(int argc, char *argv[])
{
    int ret_code;
        MM_typecode matcode;
        FILE *f;
        int M, N, nz;   
        int i, *I, *J;

        if (argc < 3)
        {
            fprintf(stderr, "Usage: %s [martix-market-filename] threadsPerBlock\n", argv[0]);
            exit(1);
        }
        else    
        { 
            if ((f = fopen(argv[1], "r")) == NULL) 
                exit(1);
        }

        if (mm_read_banner(f, &matcode) != 0)
        {
            printf("Could not process Matrix Market banner.\n");
            exit(1);
        }

        if (mm_is_complex(matcode) && mm_is_matrix(matcode) && 
                mm_is_sparse(matcode) )
        {
            printf("Sorry, this application does not support ");
            printf("Market Market type: [%s]\n", mm_typecode_to_str(matcode));
            exit(1);
        }

        /* find out size of sparse matrix .... */

        if ((ret_code = mm_read_mtx_crd_size(f, &M, &N, &nz)) !=0)
            exit(1);

    /* reseve memory for matrices */

    I = (int *) malloc(nz * sizeof(int));
    J = (int *) malloc(nz * sizeof(int));
  

    for (i=0; i<nz; i++)
    {
        fscanf(f, "%d %d\n", &I[i], &J[i]);
        I[i]--;  /* adjust from 1-based to 0-based */
        J[i]--;
    }

    if (f !=stdin) fclose(f);

    //mm_write_banner(stdout, matcode);
    //printf("nz=%d M=%d N=%d\n",nz,M,N);
    
    int* dI;
    int* dJ;
    int* col_ptr;
    int* out;
    CUDA_CALL(hipMalloc(&dI, nz*sizeof(int)));
    CUDA_CALL(hipMalloc(&dJ, nz*sizeof(int)));
    CUDA_CALL(hipMalloc(&col_ptr, N*sizeof(int)));
    CUDA_CALL(hipMalloc(&out, nz*sizeof(int)));


    hipMemcpy(dI, I, nz*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dJ, J, nz*sizeof(int), hipMemcpyHostToDevice);


    int threadsPerBlock=atoi(argv[2]);
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    //CUDA_CALL(hipMemset(col_ptr, -1, N* (sizeof(int))));

    InitColStart<<<ceil(N/threadsPerBlock), threadsPerBlock>>>(N,col_ptr);
    findColStart<<<ceil(nz/threadsPerBlock), threadsPerBlock>>>(dJ,nz,col_ptr);
    //col_ptrLengths<<<ceil(N/threadsPerBlock), threadsPerBlock>>>(N,col_ptr);


    computeRow1<<<ceil(N/64), 64,threadsPerBlock>>>(dI,dJ,nz,col_ptr,out,N);
      
    thrust::device_ptr<int> outptr(out);
    int tot = thrust::reduce(outptr, outptr + N); 
    

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf(" %f  ms ", time);





    printf(" Triangles= %d\n",tot );





    CUDA_CALL(hipFree(out));
    CUDA_CALL(hipFree(dI));
    CUDA_CALL(hipFree(dJ));
    CUDA_CALL(hipFree(col));


    return 0;
}

