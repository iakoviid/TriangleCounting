#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "mmio.h"
#include "mmio.c"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/random.h>
#include <thrust/sort.h>
#include <time.h>


#define CUDA_CALL(x)                                                           \
  {                                                                            \
    if ((x) != hipSuccess) {                                                  \
      printf("CUDA error at %s:%d\n", __FILE__, __LINE__);                     \
      printf("  %s\n", hipGetErrorString(hipGetLastError()));                \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }

__global__ void findColStart(int *dJ, int nz, int *col) {
  
 for(int i = blockIdx.x * blockDim.x + threadIdx.x+1; i<nz;i+=gridDim.x*blockDim.x){
     if(i<nz){
    int a=dJ[i];
    int b=dJ[i-1];
    if (a != b) {
      col[a] = i;
    
    if(b+1!=a){
        col[b + 1] = i;
    }}
    if(i==nz-1){
        col[a+1]=nz;
    }
    if(i==1){
    col[0]=0;

  }
  }
  } 
}


__global__ void InitColStart(int len, int *col) {
  for(int i = blockIdx.x * blockDim.x + threadIdx.x; i<len;i+=gridDim.x*blockDim.x){
  if(i<len){
        col[i]=-1;

  }}
}

__global__ void compute(int* dI,int* dJ,int nz,int* col,int* out) {
 for(int i = blockIdx.x * blockDim.x + threadIdx.x; i<nz;i+=gridDim.x*blockDim.x){

  if(i<nz){
    int s=0;
    int x=dI[i];
    int y=dJ[i];
    int k1=col[x];
    int k2=col[y];
    int r1;
    int r2;
    if(k1>0){
    int len1=col[x+1];
    int len2=col[y+1];
    while(k1<len1 && k2<len2 ) {
        if(k1>=nz || k2>=nz ){break;}
        r1=dI[k1];
        r2=dI[k2];
        if(r1==r2){
            s++;      
            k1++;
            k2++;
        }else if(r1>r2){
            k2++;

        }else{
            k1++;
        }
        

    }}



    out[i]=s;
  }}
}
int main(int argc, char *argv[])
{
    int ret_code;
        MM_typecode matcode;
        FILE *f;
        int M, N, nz;   
        int i, *I, *J;

        if (argc < 3)
    	{
    		fprintf(stderr, "Usage: %s [martix-market-filename] threadsPerBlock\n", argv[0]);
    		exit(1);
    	}
        else    
        { 
            if ((f = fopen(argv[1], "r")) == NULL) 
                exit(1);
        }

        if (mm_read_banner(f, &matcode) != 0)
        {
            printf("Could not process Matrix Market banner.\n");
            exit(1);
        }

        if (mm_is_complex(matcode) && mm_is_matrix(matcode) && 
                mm_is_sparse(matcode) )
        {
            printf("Sorry, this application does not support ");
            printf("Market Market type: [%s]\n", mm_typecode_to_str(matcode));
            exit(1);
        }

        /* find out size of sparse matrix .... */

        if ((ret_code = mm_read_mtx_crd_size(f, &M, &N, &nz)) !=0)
            exit(1);

    /* reseve memory for matrices */

    I = (int *) malloc(nz * sizeof(int));
    J = (int *) malloc(nz * sizeof(int));
  

    for (i=0; i<nz; i++)
    {
        fscanf(f, "%d %d\n", &I[i], &J[i]);
        I[i]--;  /* adjust from 1-based to 0-based */
        J[i]--;
    }

    if (f !=stdin) fclose(f);

    //mm_write_banner(stdout, matcode);
    //printf("nz=%d M=%d N=%d\n",nz,M,N);
    
    int* dI;
    int* dJ;
    int* col;
    int* out;
    CUDA_CALL(hipMalloc(&dI, nz*sizeof(int)));
    CUDA_CALL(hipMalloc(&dJ, nz*sizeof(int)));
    CUDA_CALL(hipMalloc(&col, N*sizeof(int)));
    CUDA_CALL(hipMalloc(&out, nz*sizeof(int)));



    hipMemcpy(dI, I, nz*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dJ, J, nz*sizeof(int), hipMemcpyHostToDevice);


    int threadsPerBlock=atoi(argv[2]);
    int Blocks=atoi(argv[3]);
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    CUDA_CALL(hipMemset(col, -1, N* (sizeof(int))));

    //InitColStart<<< Blocks,threadsPerBlock>>>(N,col);
    findColStart<<< Blocks,threadsPerBlock>>>(dJ,nz,col);


    compute<<<Blocks,threadsPerBlock>>>(dI,dJ,nz,col,out);
      
    thrust::device_ptr<int> outptr(out);
    int tot = thrust::reduce(outptr, outptr + nz); 
    

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("Time for Counting triangles  %f  ms \n", time);





    printf("The sum is  %d\n",tot );





    CUDA_CALL(hipFree(out));
    CUDA_CALL(hipFree(dI));
    CUDA_CALL(hipFree(dJ));
    CUDA_CALL(hipFree(col));


	return 0;
}

