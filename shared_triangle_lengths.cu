#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "mmio.h"
#include "mmio.c"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/random.h>
#include <thrust/sort.h>
#include <time.h>


#define CUDA_CALL(x)                                                           \
  {                                                                            \
    if ((x) != hipSuccess) {                                                  \
      printf("CUDA error at %s:%d\n", __FILE__, __LINE__);                     \
      printf("  %s\n", hipGetErrorString(hipGetLastError()));                \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }

__global__ void findCol_ptr(int *dJ, int nz, int *col) {
  
 for(int i = blockIdx.x * blockDim.x + threadIdx.x+1; i<nz;i+=gridDim.x*blockDim.x){
     if(i<nz){
    int x=dJ[i];
    int y=dJ[i-1];
    if (x != y) {
      col[x] = i;
    
    if(y+1!=x){
        col[y + 1] = i;
    }}
    if(i==nz-1){
        col[x+1]=nz;
    }
    if(i==0){
    col[0]=0;

  }
  }
  } 
}
__global__ void InitCol_ptr(int len, int *col) {
  for(int i = blockIdx.x * blockDim.x + threadIdx.x+1; i<len;i+=gridDim.x*blockDim.x){
  if(i<len){
        col[i]=-1;

  }}
}

#define sharedsize 65

__global__ void computeRow2(int* dI,int* dJ,int nz,int* col,int* out, int N) {
    int i=blockIdx.x;
    while(i<N){
        //if(threadIdx.x==0 && blockIdx.x==0){
        //printf("blockIdx=%d\n",blockIdx.x );}
        __shared__ int blockCol[sharedsize];//len of column
        __shared__ int nt[sharedsize];  
        int colStart=col[i];
        int len;
        if(i<N-1){
        len= col[i+1]-col[i];}
        else{len= 0;}
        int tid=threadIdx.x;
        if(colStart<0 || len==0){
            if(tid<32){
            out[i]=0;}
            return;
        }

      
        for(int j=tid;j<len;j+=blockDim.x)
        {   
            if(dJ[j+colStart]==i){
                blockCol[j]=dI[j+colStart];
                }
        }
          __syncthreads();

      
         int k1;
         int k2;
         int s;
        for(int j=tid;j<len;j+=blockDim.x)
        {   s=0;
            k1=0;
            int x=blockCol[j];
            k2=col[x];
            int r1;
            int r2;
            if(k2>0){
                int len2=col[x+1];
                r1=blockCol[k1];
                r2=dI[k2];
                while(k1<len && k2<len2 ) {
                //    if(threadIdx.x==0&& i ==0){
                //       printf("r1=%d,r2=%d\n",r1,r2 );
                //        printf("k1=%d,k2=%d\n",k1,k2 );
                //    }
                    if(r1==r2){
                        s++;      
                        k1++;
                        k2++;

                    }else if(r1>r2){
                        k2++;

                    }else{
                    k1++;
                    }
                    if(k2==nz){break;}
                    r1=blockCol[k1];
                    r2=dI[k2];
                }
            }

            nt[j]=s;
        }
        __syncthreads();
        
        for(int j=tid+blockDim.x;j<len;j+=blockDim.x){
        nt[tid]=nt[tid]+nt[j];
        
        }
        __syncthreads();


        //do reduction in shared mem 
        for( s=blockDim.x/2; s>0;s>>=1){
            if(tid<s){
            nt[tid]+=nt[tid+s];
            }
        __syncthreads();
        }

        if(tid<32){
        out[i]=nt[0];}
      
        i+=gridDim.x;
      }
}




int main(int argc, char *argv[])
{
    int ret_code;
        MM_typecode matcode;
        FILE *f;
        int M, N, nz;   
        int i, *I, *J;

        if (argc < 3)
        {
            fprintf(stderr, "Usage: %s [martix-market-filename] threadsPerBlock\n", argv[0]);
            exit(1);
        }
        else    
        { 
            if ((f = fopen(argv[1], "r")) == NULL) 
                exit(1);
        }

        if (mm_read_banner(f, &matcode) != 0)
        {
            printf("Could not process Matrix Market banner.\n");
            exit(1);
        }

        if (mm_is_complex(matcode) && mm_is_matrix(matcode) && 
                mm_is_sparse(matcode) )
        {
            printf("Sorry, this application does not support ");
            printf("Market Market type: [%s]\n", mm_typecode_to_str(matcode));
            exit(1);
        }

        /* find out size of sparse matrix .... */

        if ((ret_code = mm_read_mtx_crd_size(f, &M, &N, &nz)) !=0)
            exit(1);

    /* reseve memory for matrices */

    I = (int *) malloc(nz * sizeof(int));
    J = (int *) malloc(nz * sizeof(int));
  

    for (i=0; i<nz; i++)
    {
        fscanf(f, "%d %d\n", &I[i], &J[i]);
        I[i]--;  /* adjust from 1-based to 0-based */
        J[i]--;
    }

    if (f !=stdin) fclose(f);

    //mm_write_banner(stdout, matcode);
    //printf("nz=%d M=%d N=%d\n",nz,M,N);
    
    int* dI;
    int* dJ;
    int* col;
    int* out;
    CUDA_CALL(hipMalloc(&dI, nz*sizeof(int)));
    CUDA_CALL(hipMalloc(&dJ, nz*sizeof(int)));
    CUDA_CALL(hipMalloc(&col, N*sizeof(int)));
    CUDA_CALL(hipMalloc(&out, nz*sizeof(int)));


    hipMemcpy(dI, I, nz*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dJ, J, nz*sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock=atoi(argv[2]);
    int Blocks=atoi(argv[3]);    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    //CUDA_CALL(hipMemset(col, -1, N* (sizeof(int))));

    InitCol_ptr<<<ceil(N/threadsPerBlock), Blocks,threadsPerBlock>>>(N,col);
    findCol_ptr<<<ceil(nz/threadsPerBlock), Blocks,threadsPerBlock>>>(dJ,nz,col);
    //colLengths<<<ceil(N/threadsPerBlock), threadsPerBlock>>>(N,col);


    computeRow2<<<ceil(N/Blocks), Blocks,threadsPerBlock>>>(dI,dJ,nz,col,out,N);
      
    thrust::device_ptr<int> outptr(out);
    int tot = thrust::reduce(outptr, outptr + N); 
    

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("%f  ms ", time);





    printf(" Trianles =  %d\n",tot );





    CUDA_CALL(hipFree(out));
    CUDA_CALL(hipFree(dI));
    CUDA_CALL(hipFree(dJ));
    CUDA_CALL(hipFree(col));


    return 0;
}

